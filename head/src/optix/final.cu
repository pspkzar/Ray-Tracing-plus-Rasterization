#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

#define COEF 1.1f
#define MAX_DEPTH 5

#define SQRT_SAMPLES 10
#define OCC_DIST 0.5
#define OCC_INTEN 1.6f

struct PerRayDataResult
{
	float3 radiance;
	int depth;
};

struct PerRayData_Shadow{
	float val;
	int hit;
};

rtDeclareVariable(float3,        eye, , );

rtDeclareVariable(float4, lightDir, , );
rtDeclareVariable(float3, lightColor, , );

rtDeclareVariable(rtObject,      top_object, , );

// Material
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(int, texCount, , );
rtTextureSampler<float4, 2> tex0;

rtBuffer<float4> vertex_buffer;     
rtBuffer<uint> index_buffer;
rtBuffer<float4> normal;
rtBuffer<float4> texCoord0;

rtTextureSampler<float4,2> pos_buffer;
rtTextureSampler<float4,2> norm_buffer;
rtTextureSampler<float4,2> color_buffer;

rtBuffer<float4,2> output0;


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(PerRayDataResult, prdr, rtPayload, );
rtDeclareVariable(PerRayData_Shadow, prds, rtPayload, );

rtDeclareVariable(float3, texCoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(int, Phong, , );
rtDeclareVariable(int, Shadow, , );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

__device__ inline float rand(float2 co){
	float intpart;
    return modf(sin(dot(co,make_float2( 12.9898, 78.233))) * 43758.5453,&intpart);
}

RT_PROGRAM void buffer_camera(void)
{

	float4 pos = tex2D( pos_buffer, launch_index.x, launch_index.y );
	float4 norm = tex2D( norm_buffer, launch_index.x, launch_index.y );
	float4 color = tex2D( color_buffer, launch_index.x, launch_index.y );
	PerRayDataResult prdr;
	PerRayData_Shadow prds;
	
	float3 color_res=make_float3(0.0);
	
	if(color.x>0.0f && color.y==0.0f && color.z>0.0 && color.w==0.0f){
		float3 dir = make_float3(pos) - eye;
		dir=normalize(dir);
		
		float3 dir_reflect=reflect(dir,make_float3(norm));
		Ray reflected = make_Ray(make_float3(pos),dir_reflect,Phong,0.0001f,RT_DEFAULT_MAX);
		prdr.depth=0;
		rtTrace(top_object,reflected,prdr);
		float3 color_reflect = prdr.radiance;
		prdr.radiance=color_reflect;

		float3 dir_refract=make_float3(1.f);
		if(refract(dir_refract,dir,make_float3(norm),COEF)){
			Ray refracted = make_Ray(make_float3(pos),dir_refract,Phong,0.001f,RT_DEFAULT_MAX);
			prdr.depth=0;
			rtTrace(top_object,refracted,prdr);
			float3 color_refract=prdr.radiance;

			float coef=1-COEF*COEF*(1-pow(dot(make_float3(norm),-dir),2));

			color_res=(coef*color_refract)+((1-coef)*color_reflect);
		}
		else{
			color_res = prdr.radiance;
		}
		

	}
	else {
		color_res=make_float3(color);

		Ray shadow=make_Ray(make_float3(pos),make_float3(-lightDir),Shadow,0.0001f,RT_DEFAULT_MAX);
		rtTrace(top_object,shadow,prds);
		color_res*=prds.val;
	}
	
	
	if(SQRT_SAMPLES){
		int result=SQRT_SAMPLES*SQRT_SAMPLES;
		Onb onb(make_float3(norm));
		float inv_sqrt_samples=1/float(SQRT_SAMPLES);
		for(int i=0; i<SQRT_SAMPLES; i++){
			for(int j=0; j<SQRT_SAMPLES; j++){
				int a=i+1, b=j+1;
				float rand1=rand(make_float2(pos.x*a,pos.y*b));
				float rand2=rand(make_float2(pos.x*b,pos.z*a));
				float rand3=rand(make_float2(pos.y*a,pos.z*b));
				float rand4=rand(make_float2(rand1*b,rand2*a));
				float rand5=rand(make_float2(rand3*a,rand4*b));

				float u1=(float(i)+rand5)*inv_sqrt_samples;
				float u2=(float(j)+rand4)*inv_sqrt_samples;

				float3 occ_dir;
				cosine_sample_hemisphere(u1, u2,occ_dir);
				onb.inverse_transform(occ_dir);
				Ray occ_ray = make_Ray(make_float3(pos),occ_dir,Shadow,0.0001f,OCC_DIST);
				rtTrace(top_object,occ_ray,prds);
				result-=prds.hit;
			}
		}
	

		float occlusion_level=float(result)/float(SQRT_SAMPLES*SQRT_SAMPLES);
		occlusion_level=pow(occlusion_level,OCC_INTEN);
		color_res*=occlusion_level;
	}

	output0[launch_index]=make_float4(color_res,1.0);
}

RT_PROGRAM void any_hit_shadow(){
	prds.val=0.5;
	prds.hit=1;
	rtTerminateRay();
}

RT_PROGRAM void any_hit_shadow_glass(){
	prds.val*=0.9;
	prds.hit=0;
	rtIgnoreIntersection();
}

RT_PROGRAM void miss_shadow(){
	prds.val=1.0;
	prds.hit=0;
}

RT_PROGRAM void closest_hit(){
	float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 position=t_hit*ray.direction+ray.origin;

	float intensity=max(dot(make_float3(-lightDir),ffnormal),0.0f);
	float3 lightInensityDiffuse=lightColor*intensity;
	float3 tmp_color=make_float3(1.f);

	if(texCount==0){
		tmp_color=make_float3(diffuse)*lightInensityDiffuse + (0.3*make_float3(diffuse));
	}
	else{
		tmp_color=(make_float3(diffuse)*lightInensityDiffuse+0.3)*make_float3(tex2D(tex0, texCoord.x, texCoord.y));
	}

	prds.val=1;
	Ray shadow=make_Ray(position,make_float3(-lightDir),Shadow,0.0001f,RT_DEFAULT_MAX);
	rtTrace(top_object,shadow,prds);
	tmp_color*=prds.val;

	if(SQRT_SAMPLES){
		int result=SQRT_SAMPLES*SQRT_SAMPLES;
		Onb onb(ffnormal);
		float inv_sqrt_samples=1/float(SQRT_SAMPLES);
		for(int i=0; i<SQRT_SAMPLES; i++){
			for(int j=0; j<SQRT_SAMPLES; j++){
				int a=i+1, b=j+1;
				float rand1=rand(make_float2(position.x*a,position.y*b));
				float rand2=rand(make_float2(position.x*b,position.z*a));
				float rand3=rand(make_float2(position.y*a,position.z*b));
				float rand4=rand(make_float2(rand1*b,rand2*a));
				float rand5=rand(make_float2(rand3*a,rand4*b));

				float u1=(float(i)+rand5)*inv_sqrt_samples;
				float u2=(float(j)+rand4)*inv_sqrt_samples;

				float3 occ_dir;
				cosine_sample_hemisphere(u1, u2,occ_dir);
				onb.inverse_transform(occ_dir);
				Ray occ_ray = make_Ray(position,occ_dir,Shadow,0.0001f,OCC_DIST);
				rtTrace(top_object,occ_ray,prds);
				result-=prds.hit;
			}
		}

		float occlusion_level=float(result)/float(SQRT_SAMPLES*SQRT_SAMPLES);
		occlusion_level=pow(occlusion_level,OCC_INTEN);
		tmp_color*=occlusion_level;
	}

	prdr.radiance=tmp_color;
}

RT_PROGRAM void closest_hit_glass(){
	if(prdr.depth<MAX_DEPTH){
		
		prdr.depth++;
		int current_depth=prdr.depth;

		float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
		float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
		float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);
		float3 position = ray.origin + t_hit * ray.direction;

		float3 dir_reflect=reflect(ray.direction,ffnormal);
		Ray refl=make_Ray(position,dir_reflect,Phong,0.0001, RT_DEFAULT_MAX);
		rtTrace(top_object,refl,prdr);
		float3 color_reflect=0.99*prdr.radiance;
		


		float3 dir_refract=make_float3(1.f);
		if(refract(dir_refract,ray.direction,ffnormal,COEF)){
			Ray refracted=make_Ray(position,dir_refract,Phong,0.0001f,RT_DEFAULT_MAX);
			prdr.depth=current_depth;
			rtTrace(top_object,refracted,prdr);
			float3 color_refract=0.99*prdr.radiance;

			float coef=1-COEF*COEF*(1-pow(dot(ffnormal,-ray.direction),2));
			prdr.radiance=color_reflect;//(coef*color_refract)+((1-coef)*color_reflect);
		}
	}
	else{
		prdr.radiance=make_float3(0.f);
	}
}

RT_PROGRAM void miss(){
	prdr.radiance=make_float3(0.0f);
}

RT_PROGRAM void exception(void)
{
	output0[launch_index] = make_float4(1.f, 0.f, 0.f, 1.f);
}


RT_PROGRAM void geometryintersection(int primIdx)
{

	float4 vecauxa = vertex_buffer[index_buffer[primIdx*3]];
	float4 vecauxb = vertex_buffer[index_buffer[primIdx*3+1]];
	float4 vecauxc = vertex_buffer[index_buffer[primIdx*3+2]];
//	float3 e1, e2, h, s, q;
//	float a,f,u,v,t;

	float3 v0 = make_float3(vecauxa);
	float3 v1 = make_float3(vecauxb);
	float3 v2 = make_float3(vecauxc);

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, v0, v1, v2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      float3 n0 = make_float3(normal[ index_buffer[primIdx*3]]);
      float3 n1 = make_float3(normal[ index_buffer[primIdx*3+1]]);
      float3 n2 = make_float3(normal[ index_buffer[primIdx*3+2]]);

	  float3 t0 = make_float3(texCoord0[ index_buffer[primIdx*3]]);
	  float3 t1 = make_float3(texCoord0[ index_buffer[primIdx*3+1]]);
	  float3 t2 = make_float3(texCoord0[ index_buffer[primIdx*3+2]]);

      shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
	  texCoord =  t0*(1.0f-beta-gamma) + t1*beta + t2*gamma ;
      geometric_normal = normalize( n );

	  rtReportIntersection(0);
    }
  }
}


RT_PROGRAM void boundingbox(int primIdx, float result[6])
{
	float3 v0 = make_float3(vertex_buffer[index_buffer[primIdx*3]]);
	float3 v1 = make_float3(vertex_buffer[index_buffer[primIdx*3+1]]);
	float3 v2 = make_float3(vertex_buffer[index_buffer[primIdx*3+2]]);  
	
	const float  area = length(cross(v1-v0, v2-v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} 
	else {
	    aabb->invalidate();
	}
}
