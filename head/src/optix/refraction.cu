#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

#define COEF 1.1f
#define MAX_DEPTH 5

struct PerRayDataResult
{
	float3 radiance;
	int depth;
};

rtDeclareVariable(float3,        eye, , );

rtDeclareVariable(float4, lightDir, , );
rtDeclareVariable(float3, lightColor, , );

rtDeclareVariable(rtObject,      top_object, , );

// Material
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(int, texCount, , );
rtTextureSampler<float4, 2> tex0;

rtBuffer<float4> vertex_buffer;     
rtBuffer<uint> index_buffer;
rtBuffer<float4> normal;
rtBuffer<float4> texCoord0;

rtTextureSampler<float4,2> pos_buffer;
rtTextureSampler<float4,2> norm_buffer;
rtTextureSampler<float4,2> color_buffer;

rtBuffer<float4,2> output0;


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(PerRayDataResult, prdr, rtPayload, );

rtDeclareVariable(float3, texCoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(int, Phong, , );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

__device__ inline float rand(float2 co){
	float intpart;
    return modf(sin(dot(co,make_float2( 12.9898, 78.233))) * 43758.5453,&intpart);
}

RT_PROGRAM void buffer_camera(void)
{

	float4 pos = tex2D( pos_buffer, launch_index.x, launch_index.y );
	float4 norm = tex2D( norm_buffer, launch_index.x, launch_index.y );
	float4 color = tex2D( color_buffer, launch_index.x, launch_index.y );
	PerRayDataResult prdr;
	
	
	
	if(color.x>0.0f && color.y==0.0f && color.z>0.0 && color.w==0.0f){
		float3 dir = make_float3(pos) - eye;
		dir=normalize(dir);
		
		float3 dir_reflect=reflect(dir,make_float3(norm));
		Ray reflected = make_Ray(make_float3(pos),dir_reflect,Phong,0.0001f,RT_DEFAULT_MAX);
		prdr.depth=0;
		rtTrace(top_object,reflected,prdr);
		float3 color_reflect = prdr.radiance;
		prdr.radiance=color_reflect;

		float3 dir_refract=make_float3(1.f);
		if(refract(dir_refract,dir,make_float3(norm),COEF)){
			Ray refracted = make_Ray(make_float3(pos),dir_refract,Phong,0.001f,RT_DEFAULT_MAX);
			prdr.depth=0;
			rtTrace(top_object,refracted,prdr);
			float3 color_refract=prdr.radiance;

			float coef=1-COEF*COEF*(1-pow(dot(make_float3(norm),-dir),2));

			output0[launch_index]=make_float4((coef*color_refract)+((1-coef)*color_reflect),1.f);
		}
		else{
			output0[launch_index] = make_float4(prdr.radiance,1.f);
		}
	}
	else output0[launch_index]=color;
}

RT_PROGRAM void closest_hit(){
	float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	
	float intensity=max(dot(make_float3(-lightDir),ffnormal),0.0f);
	float3 lightInensityDiffuse=lightColor*intensity;
	float3 tmp_color=make_float3(1.f);

	if(texCount==0){
		tmp_color=make_float3(diffuse)*lightInensityDiffuse + (0.3*make_float3(diffuse));
	}
	else{
		tmp_color=(make_float3(diffuse)*lightInensityDiffuse+0.3)*make_float3(tex2D(tex0, texCoord.x, texCoord.y));
	}
	prdr.radiance=tmp_color;
}

RT_PROGRAM void closest_hit_glass(){
	if(prdr.depth<MAX_DEPTH){
		
		prdr.depth++;
		int current_depth=prdr.depth;

		float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
		float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
		float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);
		float3 position = ray.origin + t_hit * ray.direction;

		float3 dir_reflect=reflect(ray.direction,ffnormal);
		Ray refl=make_Ray(position,dir_reflect,Phong,0.0001, RT_DEFAULT_MAX);
		rtTrace(top_object,refl,prdr);
		float3 color_reflect=0.99*prdr.radiance;
		


		float3 dir_refract=make_float3(1.f);
		if(refract(dir_refract,ray.direction,ffnormal,COEF)){
			Ray refracted=make_Ray(position,dir_refract,Phong,0.0001f,RT_DEFAULT_MAX);
			prdr.depth=current_depth;
			rtTrace(top_object,refracted,prdr);
			float3 color_refract=0.99*prdr.radiance;

			float coef=1-COEF*COEF*(1-pow(dot(ffnormal,-ray.direction),2));
			prdr.radiance=color_reflect;//(coef*color_refract)+((1-coef)*color_reflect);
		}
	}
	else{
		prdr.radiance=make_float3(0.f);
	}
}

RT_PROGRAM void miss(){
	prdr.radiance=make_float3(0.0f);
}

RT_PROGRAM void exception(void)
{
	output0[launch_index] = make_float4(1.f, 0.f, 0.f, 1.f);
}


RT_PROGRAM void geometryintersection(int primIdx)
{

	float4 vecauxa = vertex_buffer[index_buffer[primIdx*3]];
	float4 vecauxb = vertex_buffer[index_buffer[primIdx*3+1]];
	float4 vecauxc = vertex_buffer[index_buffer[primIdx*3+2]];
//	float3 e1, e2, h, s, q;
//	float a,f,u,v,t;

	float3 v0 = make_float3(vecauxa);
	float3 v1 = make_float3(vecauxb);
	float3 v2 = make_float3(vecauxc);

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, v0, v1, v2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      float3 n0 = make_float3(normal[ index_buffer[primIdx*3]]);
      float3 n1 = make_float3(normal[ index_buffer[primIdx*3+1]]);
      float3 n2 = make_float3(normal[ index_buffer[primIdx*3+2]]);

	  float3 t0 = make_float3(texCoord0[ index_buffer[primIdx*3]]);
	  float3 t1 = make_float3(texCoord0[ index_buffer[primIdx*3+1]]);
	  float3 t2 = make_float3(texCoord0[ index_buffer[primIdx*3+2]]);

      shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
	  texCoord =  t0*(1.0f-beta-gamma) + t1*beta + t2*gamma ;
      geometric_normal = normalize( n );

	  rtReportIntersection(0);
    }
  }
}


RT_PROGRAM void boundingbox(int primIdx, float result[6])
{
	float3 v0 = make_float3(vertex_buffer[index_buffer[primIdx*3]]);
	float3 v1 = make_float3(vertex_buffer[index_buffer[primIdx*3+1]]);
	float3 v2 = make_float3(vertex_buffer[index_buffer[primIdx*3+2]]);  
	
	const float  area = length(cross(v1-v0, v2-v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} 
	else {
	    aabb->invalidate();
	}
}
