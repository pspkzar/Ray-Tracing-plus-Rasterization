#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;


#define SQRT_SAMPLES 10

#define OCC_DIST 0.5

#define OCC_INTEN 1.6f

struct PerRayDataResult
{
  int occlusion;
};


//rtDeclareVariable(float4, lightDir, , );

rtDeclareVariable(rtObject,      top_object, , );

rtBuffer<float4> vertex_buffer;     
rtBuffer<uint> index_buffer;

rtTextureSampler<float4,2> pos_buffer;
rtTextureSampler<float4,2> norm_buffer;

rtBuffer<float4,2> output0;


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(PerRayDataResult, prdr, rtPayload, );

__device__ inline float rand(float2 co){
	float intpart;
    return modf(sin(dot(co,make_float2( 12.9898, 78.233))) * 43758.5453,&intpart);
}

RT_PROGRAM void buffer_camera(void)
{

	float4 pos = tex2D( pos_buffer, launch_index.x, launch_index.y );
	float4 norm = tex2D( norm_buffer, launch_index.x, launch_index.y );
	PerRayDataResult prdr;	
	int result=SQRT_SAMPLES*SQRT_SAMPLES;
	prdr.occlusion = 0;
	float3 ffnormal =make_float3(norm);// faceforward(make_float3(norm),make_float3(pos),make_float3(norm));
	//if (pos.w > 0.0f) {	
		Onb onb(ffnormal);
		float3 ray_origin = make_float3(pos);
		float3 dir;
		float inv_sqrt_samples=1/float(SQRT_SAMPLES);
		for(int i=0; i<SQRT_SAMPLES; i++){
			for(int j=0; j<SQRT_SAMPLES; j++){
				int a=i+1, b=j+1;
				float rand1=rand(make_float2(pos.x*a,pos.y*b));
				float rand2=rand(make_float2(pos.x*b,pos.z*a));
				float rand3=rand(make_float2(pos.y*a,pos.z*b));
				float rand4=rand(make_float2(rand1*b,rand2*a));
				float rand5=rand(make_float2(rand3*a,rand4*b));

				float u1=(float(i)+rand5)*inv_sqrt_samples;
				float u2=(float(j)+rand4)*inv_sqrt_samples;
				cosine_sample_hemisphere(u1, u2,dir);
				onb.inverse_transform(dir);
				optix::Ray ray = optix::make_Ray(ray_origin, dir, 0, 0.0001f, OCC_DIST);
				rtTrace(top_object, ray, prdr);
				result-=prdr.occlusion;
			}
		}
		
	/*}
	else
		result=0;*/

		float4 out=make_float4(pow(float(result)/float(SQRT_SAMPLES*SQRT_SAMPLES),OCC_INTEN));
	//prdr.result.x = ray_origin.x/256.0; prdr.result.y = ray_origin.y/256.0; prdr.result.z = ray_origin.z/256.0; prdr.result.w = 1.0;
	output0[launch_index] = out;

}


RT_PROGRAM void any_hit_shadow()
{
	prdr.occlusion =  1;
	rtTerminateRay();
}


RT_PROGRAM void miss()
{
	prdr.occlusion=0;
}

RT_PROGRAM void exception(void)
{
	output0[launch_index] = make_float4(1.f, 0.f, 0.f, 1.f);
}


RT_PROGRAM void geometryintersection(int primIdx)
{

	float4 vecauxa = vertex_buffer[index_buffer[primIdx*3]];
	float4 vecauxb = vertex_buffer[index_buffer[primIdx*3+1]];
	float4 vecauxc = vertex_buffer[index_buffer[primIdx*3+2]];
//	float3 e1, e2, h, s, q;
//	float a,f,u,v,t;

	float3 v0 = make_float3(vecauxa);
	float3 v1 = make_float3(vecauxb);
	float3 v2 = make_float3(vecauxc);

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, v0, v1, v2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

 /*     float3 n0 = make_float3(normal[ index_buffer[primIdx*3]]);
      float3 n1 = make_float3(normal[ index_buffer[primIdx*3+1]]);
      float3 n2 = make_float3(normal[ index_buffer[primIdx*3+2]]);

	  float3 t0 = make_float3(texCoord0[ index_buffer[primIdx*3]]);
	  float3 t1 = make_float3(texCoord0[ index_buffer[primIdx*3+1]]);
	  float3 t2 = make_float3(texCoord0[ index_buffer[primIdx*3+2]]);

      shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
	  texCoord =  t0*(1.0f-beta-gamma) + t1*beta + t2*gamma ;
      geometric_normal = normalize( n );*/

	  rtReportIntersection(0);
    }
  }
}


RT_PROGRAM void boundingbox(int primIdx, float result[6])
{
	float3 v0 = make_float3(vertex_buffer[index_buffer[primIdx*3]]);
	float3 v1 = make_float3(vertex_buffer[index_buffer[primIdx*3+1]]);
	float3 v2 = make_float3(vertex_buffer[index_buffer[primIdx*3+2]]);  
	
	const float  area = length(cross(v1-v0, v2-v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} 
	else {
	    aabb->invalidate();
	}
}
