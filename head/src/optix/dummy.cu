#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

#define COEF 1.2f
#define MAX_DEPTH 50

#define SQRT_SAMPLES 0
#define OCC_DIST 0.5
#define OCC_INTEN 1.6f


rtDeclareVariable(float3,        eye, , );

rtDeclareVariable(float4, lightDir, , );
rtDeclareVariable(float3, lightColor, , );

rtDeclareVariable(rtObject,      top_object, , );

// Material
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(int, texCount, , );
rtTextureSampler<float4, 2> tex0;

rtBuffer<float4> vertex_buffer;     
rtBuffer<uint> index_buffer;
rtBuffer<float4> normal;
rtBuffer<float4> texCoord0;

rtTextureSampler<float4,2> pos_buffer;
rtTextureSampler<float4,2> norm_buffer;
rtTextureSampler<float4,2> color_buffer;

rtBuffer<float4,2> output0;


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float3, texCoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(int, Phong, , );
rtDeclareVariable(int, Shadow, , );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

__device__ inline float rand(float2 co){
	float intpart;
    return modf(sin(dot(co,make_float2( 12.9898, 78.233))) * 43758.5453,&intpart);
}

RT_PROGRAM void buffer_camera(void)
{

	float4 pos = tex2D( pos_buffer, launch_index.x, launch_index.y );
	float4 norm = tex2D( norm_buffer, launch_index.x, launch_index.y );
	float4 color = tex2D( color_buffer, launch_index.x, launch_index.y );
	
	
	
	output0[launch_index]=color;
}



RT_PROGRAM void exception(void)
{
	output0[launch_index] = make_float4(1.f, 0.f, 0.f, 1.f);
}


RT_PROGRAM void geometryintersection(int primIdx)
{

	float4 vecauxa = vertex_buffer[index_buffer[primIdx*3]];
	float4 vecauxb = vertex_buffer[index_buffer[primIdx*3+1]];
	float4 vecauxc = vertex_buffer[index_buffer[primIdx*3+2]];
//	float3 e1, e2, h, s, q;
//	float a,f,u,v,t;

	float3 v0 = make_float3(vecauxa);
	float3 v1 = make_float3(vecauxb);
	float3 v2 = make_float3(vecauxc);

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, v0, v1, v2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      float3 n0 = make_float3(normal[ index_buffer[primIdx*3]]);
      float3 n1 = make_float3(normal[ index_buffer[primIdx*3+1]]);
      float3 n2 = make_float3(normal[ index_buffer[primIdx*3+2]]);

	  float3 t0 = make_float3(texCoord0[ index_buffer[primIdx*3]]);
	  float3 t1 = make_float3(texCoord0[ index_buffer[primIdx*3+1]]);
	  float3 t2 = make_float3(texCoord0[ index_buffer[primIdx*3+2]]);

      shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
	  texCoord =  t0*(1.0f-beta-gamma) + t1*beta + t2*gamma ;
      geometric_normal = normalize( n );

	  rtReportIntersection(0);
    }
  }
}


RT_PROGRAM void boundingbox(int primIdx, float result[6])
{
	float3 v0 = make_float3(vertex_buffer[index_buffer[primIdx*3]]);
	float3 v1 = make_float3(vertex_buffer[index_buffer[primIdx*3+1]]);
	float3 v2 = make_float3(vertex_buffer[index_buffer[primIdx*3+2]]);  
	
	const float  area = length(cross(v1-v0, v2-v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} 
	else {
	    aabb->invalidate();
	}
}
