#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

#define MAX_DEPTH 5

struct PerRayDataResult
{
	float3 radiance;
	int depth;
};

rtDeclareVariable(float3,        eye, , );

rtDeclareVariable(float4, lightDir, , );
rtDeclareVariable(float3, lightColor, , );

rtDeclareVariable(rtObject,      top_object, , );

// Material
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(int, texCount, , );
rtTextureSampler<float4, 2> tex0;

rtBuffer<float4> vertex_buffer;     
rtBuffer<uint> index_buffer;
rtBuffer<float4> normal;
rtBuffer<float4> texCoord0;

rtTextureSampler<float4,2> pos_buffer;
rtTextureSampler<float4,2> norm_buffer;
rtTextureSampler<float4,2> color_buffer;

rtBuffer<float4,2> output0;


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(PerRayDataResult, prdr, rtPayload, );

rtDeclareVariable(float3, texCoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(int, Phong, , );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

__device__ inline float rand(float2 co){
	float intpart;
    return modf(sin(dot(co,make_float2( 12.9898, 78.233))) * 43758.5453,&intpart);
}

RT_PROGRAM void buffer_camera(void)
{

	float4 pos = tex2D( pos_buffer, launch_index.x, launch_index.y );
	float4 norm = tex2D( norm_buffer, launch_index.x, launch_index.y );
	float4 color = tex2D( color_buffer, launch_index.x, launch_index.y );
	PerRayDataResult prdr;
	
	prdr.depth=0;
	
	if(color.x>0.0f && color.y==0.0f && color.z>0.0 && color.w==0.0f){
		float3 dir=make_float3(pos) - eye;
		dir=normalize(dir);
		float3 dir_reflect=reflect(dir,make_float3(norm));
		Ray ray = make_Ray(make_float3(pos),dir_reflect,Phong,0.0001f,RT_DEFAULT_MAX);
		rtTrace(top_object,ray,prdr);
		output0[launch_index] = make_float4(prdr.radiance,1.f);//(1.f-COEF)*color+COEF*prdr.radiance;
	}
	else output0[launch_index]=color;
}

RT_PROGRAM void closest_hit(){
	float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	
	float intensity=max(dot(make_float3(-lightDir),ffnormal),0.0f);
	float3 lightInensityDiffuse=lightColor*intensity;
	float3 tmp_color=make_float3(1.f);

	if(texCount==0){
		tmp_color=make_float3(diffuse)*lightInensityDiffuse + (0.3*make_float3(diffuse));
	}
	else{
		tmp_color=(make_float3(diffuse)*lightInensityDiffuse+0.3)*make_float3(tex2D(tex0, texCoord.x, texCoord.y));
	}
	prdr.radiance=tmp_color;
}

RT_PROGRAM void closest_hit_glass(){
	if(prdr.depth<MAX_DEPTH){
		prdr.depth++;
		float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
		float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
		float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);
		float3 dir=reflect(ray.direction,ffnormal);
		float3 position = ray.origin + t_hit * ray.direction;
		Ray refl=make_Ray(position,dir,Phong,0.0001, RT_DEFAULT_MAX);
		rtTrace(top_object,refl,prdr);
	}
	else{
		prdr.radiance=make_float3(0.f);
	}
}

RT_PROGRAM void miss(){
	prdr.radiance=make_float3(0.0f);
}

RT_PROGRAM void exception(void)
{
	output0[launch_index] = make_float4(1.f, 0.f, 0.f, 1.f);
}


RT_PROGRAM void geometryintersection(int primIdx)
{

	float4 vecauxa = vertex_buffer[index_buffer[primIdx*3]];
	float4 vecauxb = vertex_buffer[index_buffer[primIdx*3+1]];
	float4 vecauxc = vertex_buffer[index_buffer[primIdx*3+2]];
//	float3 e1, e2, h, s, q;
//	float a,f,u,v,t;

	float3 v0 = make_float3(vecauxa);
	float3 v1 = make_float3(vecauxb);
	float3 v2 = make_float3(vecauxc);

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, v0, v1, v2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      float3 n0 = make_float3(normal[ index_buffer[primIdx*3]]);
      float3 n1 = make_float3(normal[ index_buffer[primIdx*3+1]]);
      float3 n2 = make_float3(normal[ index_buffer[primIdx*3+2]]);

	  float3 t0 = make_float3(texCoord0[ index_buffer[primIdx*3]]);
	  float3 t1 = make_float3(texCoord0[ index_buffer[primIdx*3+1]]);
	  float3 t2 = make_float3(texCoord0[ index_buffer[primIdx*3+2]]);

      shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
	  texCoord =  t0*(1.0f-beta-gamma) + t1*beta + t2*gamma ;
      geometric_normal = normalize( n );

	  rtReportIntersection(0);
    }
  }
}


RT_PROGRAM void boundingbox(int primIdx, float result[6])
{
	float3 v0 = make_float3(vertex_buffer[index_buffer[primIdx*3]]);
	float3 v1 = make_float3(vertex_buffer[index_buffer[primIdx*3+1]]);
	float3 v2 = make_float3(vertex_buffer[index_buffer[primIdx*3+2]]);  
	
	const float  area = length(cross(v1-v0, v2-v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} 
	else {
	    aabb->invalidate();
	}
}
